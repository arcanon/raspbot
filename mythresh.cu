#include "hip/hip_runtime.h"
#include <opencv2/gpu/gpu.hpp>
#include <opencv2\core\cuda\common.hpp>

using namespace cv::gpu::cudev;

__global__   void rgbnorm_dev(float4* image, int pitch)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y; 

    if (x >= 512 || y >= 512)
        return;

    int linearPos = y*pitch + x*4;

    uchar *pixel = ((uchar *)(((char *)image) +linearPos));

    uchar r = pixel[0];
    uchar g = pixel[1];
    uchar b = pixel[2];

    uint sum = r + g + b;

    pixel[0] = (sum > 5 ? (uchar)((double)r/sum*255) : 0);
    pixel[1] = (sum > 5 ? (uchar)((double)g/sum*255) : 0);
    pixel[2] = (sum > 5 ? (uchar)((double)b/sum*255) : 0);
}

void rgbnorm(cv::gpu::GpuMat& image)
{
	dim3 threads(32, 8);
    dim3 grid (divUp (image.cols, threads.x), divUp (image.rows, threads.y));
    
    rgbnorm_dev<<<grid, threads>>>((float4 *)image.ptr(), image.step);

    hipError_t err ;
    err = hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        fprintf(stdout, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    }
}

__global__   void mythresh_dev(float4* src, uchar* dst, int pitch, int dstPitch, int*  bounds)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y; 

    if (x >= 512 || y >= 512)
        return;

    int linearPos = y*pitch + x*4;
    int linearPosdst = y*dstPitch + x;
    bool canditate = true;
    uchar *pixel = ((uchar *)(((char *)src) +linearPos));

    int one = bounds[0];
    if (bounds[0] > pixel[0])
        canditate = false;
    if (bounds[1] < pixel[0])
        canditate = false;
    if (bounds[2] > pixel[1])
        canditate = false;
    if (bounds[3] < pixel[1])
        canditate = false;
    if (bounds[4] > pixel[2])
        canditate = false;
    if (bounds[5] < pixel[2])
        canditate = false;
   
    //if (pixel[1] > 50) {
    //    if ((pixel[2] >= pixel[1]-15) && (pixel[2] < pixel[1]+15)) {
    //        canditate = true;
    //        
    //    }

    //   /* if ((pixel[2] >= pixel[0]-40) && (pixel[2] < pixel[0]+40)) {
    //            canditate = true;
    //        }*/
    //}
    dst[linearPosdst] = (canditate ? 255 : 0);

    //dst[linearPosdst] = __float2int_rd(0.5*255);//(((float4 *)(((char *)src) +linearPos))->x*255);
    //else
    //    dst[linearPosdst] = 125;
}


// PtrStepSzb dst
// (PtrStepSz<T>)src

void mythresh(cv::gpu::GpuMat& src, cv::gpu::GpuMat& dst, int bounds[3][2])
{
	dim3 threads(32, 8);
    dim3 grid (divUp (src.cols, threads.x), divUp (src.rows, threads.y));
    hipError_t err ;
    static int* bounds_arr = NULL;
    if (!bounds_arr) {
        err = hipMalloc(&bounds_arr,3*2*sizeof(int));
        int data[6];
        memcpy(data,&bounds[0][0],6*4); 
        
    }

    err = hipMemcpy(bounds_arr, &bounds[0][0],3*2*sizeof(int),hipMemcpyHostToDevice);

	mythresh_dev<<<grid, threads>>>((float4 *)src.ptr(), dst.ptr(), src.step, dst.step, bounds_arr);


    err = hipDeviceSynchronize();



    if (err != hipSuccess)
    {
        fprintf(stdout, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    }
}