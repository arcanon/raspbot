#include "hip/hip_runtime.h"
#include <opencv2/gpu/gpu.hpp>
#include <opencv2\core\cuda\common.hpp>

using namespace cv::gpu::cudev;

__global__   void anaglyph_dev(char* imageLeft, char* imageRight, char *imageOut, int pitchInputs, int pitchOutput, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y; 

    if (x >= width || y >= height)
        return;

    int linearPosInputs = y*pitchInputs + x;
    int linearPosOutput = y*pitchOutput + x*4;

    // Red
    imageOut[linearPosOutput]   = imageLeft[linearPosInputs];
    imageOut[linearPosOutput+1] = 0;//imageRight[linearPosInputs];
    imageOut[linearPosOutput+2] = imageRight[linearPosInputs];
}

void anaglyph(cv::gpu::GpuMat& leftImage, cv::gpu::GpuMat& rightImage, cv::gpu::GpuMat& outputImage)
{
	dim3 threads(32, 8);
    dim3 grid (divUp (leftImage.cols, threads.x), divUp (leftImage.rows, threads.y));
    
    anaglyph_dev<<<grid, threads>>>((char *)leftImage.ptr(), (char *)rightImage.ptr(), (char *)outputImage.ptr(), leftImage.step, outputImage.step, leftImage.cols, leftImage.rows);

    hipError_t err ;
    err = hipDeviceSynchronize();

    if (err != hipSuccess)
    {
        fprintf(stdout, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
    }
}
